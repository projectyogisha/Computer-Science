#include "hip/hip_runtime.h"
// cuda Matrix Addition using A[i][j] and (*a)[j] declaration and 1 dimension blocks x . 
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

const int M = 10, N = 8;

hipError_t addWithCuda(float A[M][N], float B[M][N], float C[M][N]);

__global__ void addKernel(float A[M][N], float B[M][N], float C[M][N])
{
    int i = threadIdx.x / N;	 // columns
	int j = threadIdx.x % N;     // rows
    C[i][j] = A[i][j] + B[i][j];
}

int main()
{
	// allocation in host
    float a[M][N];
	float b[M][N];
	float c[M][N];

	// initialization original data in host
	for(int i = 0; i < M; i++) {
		for(int j = 0; j < N; j++) {
			a[i][j] = i;
			b[i][j] = j;	
		}
	}

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(a, b, c);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // display the result
	for(int i = 0; i < M; i++) {
		for(int j = 0; j < N; j++) {
			printf(" %1.f ", a[i][j]);
		}
		printf("\n");
	}
	
	printf("\t Plus \t\n");

	for(int i = 0; i < M; i++) {
		for(int j = 0; j < N; j++) {
			printf(" %1.f ", b[i][j]);
		}
		printf("\n");
	}

	printf("========================\n");

	for(int i = 0; i < M; i++) {
		for(int j = 0; j < N; j++) {
			printf(" %1.f ", c[i][j]);
		}
		printf("\n");
	}

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(float A[M][N], float B[M][N], float C[M][N])
{
    // create device varibles
	float (*d_a)[N];
    float (*d_b)[N];
    float (*d_c)[N];
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&d_a, M * N * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&d_b, M * N * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&d_c, M * N * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(d_a, A, M * N * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(d_b, B, M * N * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
	dim3 BlocksPerGrid(1, 1, 1);
	dim3 ThreadsPerBlock(M * N, 1, 1);
    addKernel<<<BlocksPerGrid, ThreadsPerBlock>>>(d_a, d_b, d_c); 

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(C, d_c, M * N * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    return cudaStatus;
}

